#include "hip/hip_runtime.h"
#ifdef _MSC_VER
#include "msvc_defines.h"
#endif
#include <hip/hip_runtime.h>
#include <iostream>
#include <locale>
#include <string>
#include <vector>

#define HIP_CHECK(status)                                                      \
  if (status != hipSuccess) {                                                  \
    fprintf(stderr, "error: '%s' at %s:%d\n", hipGetErrorString(status),       \
            __FILE__, __LINE__);                                               \
    exit(0);                                                                   \
  }

__global__ void kernel(const wchar_t *in, wchar_t *out, const size_t dim,
                       const wchar_t mark) {
  const int idx = hipThreadIdx_x * dim + hipThreadIdx_y;
  if (hipBlockIdx_x) {
    out[idx * 2 + 1] = mark;
  } else {
    out[idx * 2] = in[idx];
  }
}

wchar_t get_mark(char *s) {
  const char a = s[0];
  const char b = s[1];
  const bool u = b != '\0' && b == 'u';
  if (a == 'q') {
    return u ? L'\x00BF' : L'\xFF1F';
  } else if (a == 'e' && u) {
    return L'\x00A1';
  }
  return L'\xFF01';
}

int main(int argc, char *argv[]) {
  std::setlocale(LC_ALL, "");
  std::vector<wchar_t> str;

  wchar_t c;
  while ((c = std::wcin.get()) != WEOF) {
    if (c == ' ') {
      continue;
    }
    str.push_back(c);
  }

  wchar_t *input;
  wchar_t *output;
  const size_t length = str.size();
  const size_t dim = ceil(sqrt(double(length)));
  const size_t input_size = sizeof(wchar_t) * dim * dim;
  const size_t output_length = dim * 2 * dim;
  const size_t output_size = sizeof(wchar_t) * output_length;
  HIP_CHECK(hipMalloc((void **)&input, input_size));
  HIP_CHECK(hipMalloc((void **)&output, output_size));
  HIP_CHECK(hipMemcpy(input, str.data(), sizeof(wchar_t) * length,
                      hipMemcpyHostToDevice));

  const wchar_t mark = argc < 2 ? L'\xFF01' : get_mark(argv[1]);
  kernel<<<2, dim3(dim, dim), 0, 0>>>(input, output, dim, mark);

  auto result = new wchar_t[output_length + 2]; // mark ... \x0000
  *result = mark;
  HIP_CHECK(hipMemcpy(result + 1, output, output_size, hipMemcpyDeviceToHost));
  result[length * 2 + 1] = 0;

  HIP_CHECK(hipFree(input));
  HIP_CHECK(hipFree(output));

  std::wcout << result << L'\n';

  delete[] result;

  return 0;
}
