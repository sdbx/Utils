#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <locale>
#include <string>
#include <vector>

#define CUDA_CHECK(status)                                                     \
  if (status != hipSuccess) {                                                 \
    fprintf(stderr, "error: '%s' at %s:%d\n", hipGetErrorString(status),      \
            __FILE__, __LINE__);                                               \
    exit(0);                                                                   \
  }

__global__ void kernel(const wchar_t *in, wchar_t *out, const size_t dim,
                       const wchar_t mark) {
  const int idx = threadIdx.x * dim + threadIdx.y;
  out[idx * 2 + blockIdx.x] = blockIdx.x == 0 ? in[idx] : mark;
}

wchar_t get_mark(char *s) {
  const char a = s[0];
  const char b = s[1];
  const bool u = b != '\0' && b == 'u';
  if (a == 'q') {
    return u ? L'\x00BF' : L'\xFF1F';
  } else if (a == 'e' && u) {
    return L'\x00A1';
  }
  return L'\xFF01';
}

int main(int argc, char *argv[]) {
  std::setlocale(LC_ALL, "");
  std::vector<wchar_t> str;

  wchar_t c;
  while ((c = std::wcin.get()) != (wchar_t)WEOF) {
    if (c == ' ') {
      continue;
    }
    str.push_back(c);
  }

  wchar_t *input;
  wchar_t *output;
  const size_t length = str.size();
  const size_t dim = ceil(sqrt(double(length)));
  const size_t input_size = sizeof(wchar_t) * dim * dim;
  const size_t output_length = dim * 2 * dim;
  const size_t output_size = sizeof(wchar_t) * output_length;
  CUDA_CHECK(hipMalloc((void **)&input, input_size));
  CUDA_CHECK(hipMalloc((void **)&output, output_size));
  CUDA_CHECK(hipMemcpy(input, str.data(), sizeof(wchar_t) * length,
                        hipMemcpyHostToDevice));

  const wchar_t mark = argc < 2 ? L'\xFF01' : get_mark(argv[1]);
  kernel<<<2, dim3(dim, dim), 0, 0>>>(input, output, dim, mark);

  auto result = new wchar_t[output_length + 2]; // mark ... \x0000
  *result = mark;
  CUDA_CHECK(
      hipMemcpy(result + 1, output, output_size, hipMemcpyDeviceToHost));
  result[length * 2 + 1] = L'\0';

  CUDA_CHECK(hipFree(input));
  CUDA_CHECK(hipFree(output));

  std::wcout << result << L'\n';

  delete[] result;

  return 0;
}
